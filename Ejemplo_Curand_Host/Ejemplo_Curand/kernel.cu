#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""
#include "hiprand.h"
#include <ctime>
#include <stdio.h>





__global__ void reduction(float* dev_rnd, float* dev_output)
{
	extern __shared__ float intermedio[];
	unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	unsigned int tid = threadIdx.x;
	intermedio[threadIdx.x] = dev_rnd[idx] + dev_rnd[idx + blockDim.x];
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s != 0; s >>= 1) {
		if (tid < s) {
			intermedio[tid] += intermedio[tid + s];
		}
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		dev_output[blockIdx.x] = intermedio[0];
	}

	
}

int RandomHostAPI_Test(int numBlocks,int numThreads) {
	float* dev_rnd;
	float* dev_output;
	hipError_t cudaStatus; 
	hiprandGenerator_t generator;
	hiprandStatus_t s  = hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW);
	if (s != HIPRAND_STATUS_SUCCESS) {
		return s; // cualquiera != 0;
	}
	s = hiprandSetPseudoRandomGeneratorSeed(generator, clock());
	int N = numBlocks * numThreads;
	cudaStatus = hipMalloc((void**)&dev_rnd, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&dev_output, numBlocks * sizeof(float));
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	s = hiprandGenerateUniform(generator, dev_rnd, N);
	if (s != HIPRAND_STATUS_SUCCESS) {
		return s; // cualquiera != 0;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	s = hiprandDestroyGenerator(generator);
	if (s != HIPRAND_STATUS_SUCCESS) {
		return s; // cualquiera != 0;
	}
	reduction << < numBlocks / 2, numThreads, numThreads * sizeof(float) >> > (dev_rnd, dev_output);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	float* host_output;
	cudaStatus = hipHostMalloc(&host_output, numBlocks * sizeof(float));
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(host_output, dev_output, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	float suma = 0;
	for (int i = 0; i < numBlocks; i++) {
		suma += host_output[i];
	}
	printf("Sum: %f", suma * 2);

	hipHostFree(host_output);
	hipFree(dev_output);
	hipFree(dev_rnd);
	return cudaStatus;
}

int main()
{


	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

    cudaStatus = (hipError_t) RandomHostAPI_Test(100000,1024);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    return 0;
}
