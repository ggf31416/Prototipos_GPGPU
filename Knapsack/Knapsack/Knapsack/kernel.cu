#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <ctime>
#include <stdio.h>
#include "notbitwise.cuh"
#include "stdint.h"
#include "ErrorInfo.h"
#include "bitwise.cuh"
#include <ctime>
#include <iostream>
#include <string>


#ifdef __INTELLISENSE__

//for __syncthreads()
#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif // !(__CUDACC_RTC__)
//for atomicAdd
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#define __DEVICE_FUNCTIONS_H__

#endif


#define makeRandomInts makeRandomIntegers2

struct EvalInfo {
	float min;
	float minValido;
	float max;
	float avg;
	float avgValido;
	float avgPenal;
	int invalidos;
};

float timeFitness, timeCross;

bool KERNEL_TIMING = false;
bool UNIT_TEST = false;
int SALIDA_STEP = 500; // cada cuanto mostrar estadisticas

template<typename T>
__global__ void sumar(T* dev_rnd, float* dev_output,unsigned int len)
{
	__shared__ float intermedio[MAX_THREADS_PER_BLOCK];
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	intermedio[threadIdx.x] = idx < len ?  dev_rnd[idx] : 0;
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s != 0; s >>= 1) {
		if (tid < s) {
			intermedio[tid] += intermedio[tid + s];
		}
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		dev_output[blockIdx.x] = intermedio[0];
	}
}

__global__ void minimo(float* dev_rnd, float* dev_output,unsigned int len)
{
	__shared__ float intermedio[MAX_THREADS_PER_BLOCK];
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	intermedio[threadIdx.x] = idx < len ? dev_rnd[idx] : dev_rnd[0];
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s != 0; s >>= 1) {
		if (tid < s) {
			intermedio[tid] = min(intermedio[tid], intermedio[tid + s]);
		}
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		dev_output[blockIdx.x] = intermedio[0];
	}
}

__global__ void maximo(float* dev_rnd, float* dev_output, unsigned int len)
{
	__shared__ float intermedio[MAX_THREADS_PER_BLOCK];
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	intermedio[threadIdx.x] = idx < len ? dev_rnd[idx] : dev_rnd[0];
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s != 0; s >>= 1) {
		if (tid < s) {
			intermedio[tid] = max(intermedio[tid], intermedio[tid + s]);
		}
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		dev_output[blockIdx.x] = intermedio[0];
	}
}

__global__ void contarInvalidos(float* fitness, float* dev_output, size_t pop_size) {
	__shared__ float intermedio[MAX_THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	intermedio[tid] = 0;
	for (int i = tid; i < pop_size; i += MAX_THREADS_PER_BLOCK) {
		intermedio[tid] += (fitness[i] < 0) ? 1 : 0;
	}
	// reduction algorithm to add the partial fitness values
	__syncthreads();
	int i = MAX_THREADS_PER_BLOCK / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			intermedio[threadIdx.x] = intermedio[threadIdx.x] + intermedio[threadIdx.x + i];
		__syncthreads();
		i = i / 2;
	}

	// finally thread 0 writes the fitness value in global memory
	if (threadIdx.x == 0)
		dev_output[blockIdx.x] = intermedio[0];
}

__global__ void sumarValidez(float* fitness, float* dev_output, size_t pop_size) {
	__shared__ float intermedio_val[MAX_THREADS_PER_BLOCK];
	__shared__ float intermedio_inval[MAX_THREADS_PER_BLOCK];
	__shared__ float intermedio_cant[MAX_THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	intermedio_val[tid] = 0;
	intermedio_inval[tid] = 0;
	intermedio_cant[tid] = 0;
	for (int i = tid; i < pop_size; i += MAX_THREADS_PER_BLOCK) {
		if (fitness[i]  >= 0 ) {
			intermedio_val[tid] +=  fitness[i];
		}
		else {
			intermedio_inval[tid] += fitness[i];
			intermedio_cant[tid] += 1;
		}
		
	}
	// reduction algorithm to add the partial fitness values
	__syncthreads();
	int i = MAX_THREADS_PER_BLOCK / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			intermedio_val[threadIdx.x] = intermedio_val[threadIdx.x] + intermedio_val[threadIdx.x + i];
		__syncthreads();
		i = i / 2;
	}

	// reduction algorithm to add the partial fitness values
	__syncthreads();
	 i = MAX_THREADS_PER_BLOCK / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			intermedio_inval[threadIdx.x] = intermedio_inval[threadIdx.x] + intermedio_inval[threadIdx.x + i];
		__syncthreads();
		i = i / 2;
	}

	// reduction algorithm to add the partial fitness values
	__syncthreads();
	 i = MAX_THREADS_PER_BLOCK / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			intermedio_cant[threadIdx.x] = intermedio_cant[threadIdx.x] + intermedio_cant[threadIdx.x + i];
		__syncthreads();
		i = i / 2;
	}

	// finally thread 0 writes the fitness value in global memory
	if (threadIdx.x == 0) {
		dev_output[0] = intermedio_cant[0]; // cant invalidos
		dev_output[1] = intermedio_inval[0]; // suma invalidos 
		dev_output[2] = intermedio_val[0]; // suma validos

	}
		
}

__global__ void minimoValidos(float* fitness, float* dev_output, size_t pop_size) {
	__shared__ float intermedio[MAX_THREADS_PER_BLOCK];
	unsigned int tid = threadIdx.x;
	intermedio[tid] = 0;
	for (int i = tid; i < pop_size; i += MAX_THREADS_PER_BLOCK) {
		if (fitness[i] > 0) {
			intermedio[tid] = intermedio[tid] > 0 ? min(intermedio[tid], fitness[i]) : fitness[i];
		}
	}
	// reduction algorithm to add the partial fitness values
	__syncthreads();
	int i = MAX_THREADS_PER_BLOCK / 2;
	while (i != 0) {
		if (threadIdx.x < i){
			if (intermedio[threadIdx.x] > 0 && intermedio[threadIdx.x + i] > 0) {
				// el menor de los valores > 0
				intermedio[threadIdx.x] = min(intermedio[threadIdx.x], intermedio[threadIdx.x + i]);
			}
			else {
				// el mayor de los dos valores (el mayor a 0 o 0 si los 2 son 0)
				intermedio[threadIdx.x] = max(intermedio[threadIdx.x], intermedio[threadIdx.x + i]);
			}
			
		}
		__syncthreads();
		i = i / 2;
	}

	// finally thread 0 writes the fitness value in global memory
	if (threadIdx.x == 0)
		dev_output[blockIdx.x] = intermedio[0];
}




__global__ void scaleRandom(float* floatRnd, int* intRnd, size_t N, unsigned int scale) {
	unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (pos < N) {
		intRnd[pos] = __float2int_rd(floatRnd[pos] * (scale + 0.999999f));
	}

}


ErrorInfo makeRandomIntegers(hiprandGenerator_t& generator, int* indices, unsigned int N, unsigned int max) {
	ErrorInfo status;
	float* rndFloat;

	status.cuda = hipMalloc(&rndFloat, N * sizeof(float));
	if (status.failed()) return status;

	status.hiprand = hiprandGenerateUniform(generator, rndFloat, N);
	status.cuda = hipDeviceSynchronize();
	if(status.failed()) return status;

	unsigned int blocks = (N + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK; // ceil(N/MAX_THREADS_PER_BLOCK)
	scaleRandom << <blocks, MAX_THREADS_PER_BLOCK >> >(rndFloat, indices, N, max);
	status.cuda = hipGetLastError();
	if (status.failed()) return status;

	status.cuda = hipDeviceSynchronize();

	hipFree(rndFloat);
	return status;

}


__global__ void scaleRandom2(uint32_t* rnd,  size_t N, double scale) {
	unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (pos < N) {
		// multiplicar aleatorio por escala, mul
		rnd[pos] = __double2uint_rd(__dmul_rd (rnd[pos] , scale));
	}

}

__global__ void scaleRandomMod(uint32_t* rnd, size_t N, uint32_t max1) {
	unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
	if (pos < N) {
		rnd[pos] = rnd[pos] % max1;
	}

}



ErrorInfo makeRandomIntegers2(hiprandGenerator_t& generator, int32_t* indices, unsigned int N, unsigned int max) {
	ErrorInfo status;
	uint32_t* uindices = (uint32_t*)indices; // reinterpreto indices como si fueran unsigned
	double scale = (double)(max + (1 - 1e-6)) / ((1LL << 32) - 1) ;

	status.hiprand = hiprandGenerate(generator, uindices, N);
	status.cuda = hipDeviceSynchronize();
	if (status.failed()) return status;

	unsigned int blocks = (N + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK; // ceil(N/MAX_THREADS_PER_BLOCK)
	scaleRandom2 << <blocks, MAX_THREADS_PER_BLOCK >> >( uindices, N, scale);
	status.cuda = hipGetLastError();
	if (status.failed()) return status;

	status.cuda = hipDeviceSynchronize();

	return status;

}

ErrorInfo makeRandomIntegersMod(hiprandGenerator_t& generator, int32_t* indices, unsigned int N, unsigned int max) {
	ErrorInfo status;
	uint32_t* uindices = (uint32_t*)indices; // reinterpreto indices como si fueran unsigned

	status.hiprand = hiprandGenerate(generator, uindices, N);
	status.cuda = hipDeviceSynchronize();
	if (status.failed()) return status;

	unsigned int blocks = (N + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK; // ceil(N/MAX_THREADS_PER_BLOCK)
	scaleRandomMod << <blocks, MAX_THREADS_PER_BLOCK >> >(uindices, N, max+1);
	status.cuda = hipGetLastError();
	if (status.failed()) return status;

	status.cuda = hipDeviceSynchronize();

	return status;

}



hiprandStatus_t initGenerator(hiprandGenerator_t& generator ,unsigned long long seed) {
	hiprandStatus_t s =  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	if (s != HIPRAND_STATUS_SUCCESS) {
		return s; 
	}
	s = hiprandSetPseudoRandomGeneratorSeed(generator,seed);
	return s;
}

ErrorInfo initProbs(float** probs, int** points, size_t POP_SIZE) {
	ErrorInfo status;
	status.cuda = hipMalloc(probs, POP_SIZE * sizeof(float));
	if (status.failed()) return status;
	status.cuda = hipMalloc(points, POP_SIZE * sizeof(int));
	return status;
}

ErrorInfo makeRandomNumbersMutation(hiprandGenerator_t& generator, size_t POP_SIZE, int len, float* randomPM, int* randomPoint) {
	ErrorInfo status; 

	status.hiprand = hiprandGenerateUniform(generator, randomPM, POP_SIZE);
	if (status.failed()) return status;

	status = makeRandomInts(generator, randomPoint, POP_SIZE, len - 1);
	if (status.failed()) return status;
	status.cuda = hipDeviceSynchronize();

	return status;

}



ErrorInfo makeRandomNumbersSpx(hiprandGenerator_t& generator, size_t POP_SIZE, int len, float* randomPC, int* randomPoint) {
	ErrorInfo status;
	size_t HALF_SIZE = POP_SIZE / 2;

	status = makeRandomInts(generator, randomPoint, HALF_SIZE, len - 1);
	if (status.failed()) return status;

	status.hiprand = hiprandGenerateUniform(generator, randomPC, HALF_SIZE);
	status.cuda = hipDeviceSynchronize();

	return status;

}

ErrorInfo makeRandomNumbersDpx(hiprandGenerator_t& generator, size_t POP_SIZE, int len, float* randomPC, int* randomPoint) {
	ErrorInfo status;
	size_t HALF_SIZE = POP_SIZE / 2;

	status = makeRandomInts(generator, randomPoint, POP_SIZE, len - 1);
	if (status.failed()) return status;

	status.hiprand = hiprandGenerateUniform(generator, randomPC, HALF_SIZE);
	status.cuda = hipDeviceSynchronize();

	return status;

}

hipError_t InitTournRandom( int** random, size_t POP_SIZE) {
	return hipMalloc(random, 2 * sizeof( int) * POP_SIZE);
}



ErrorInfo makeRandomNumbersTournement(hiprandGenerator_t& generator, size_t POP_SIZE, int* random) {
	// generar (POPSIZE * 2) numeros aleatorios enteros de 0 a POPSIZE - 1
	return makeRandomInts(generator, random, POP_SIZE * 2, POP_SIZE - 1);
}

hipError_t InitFit(float** dev_fit,size_t POP_SIZE) {
	return hipMalloc(dev_fit, sizeof(float) * POP_SIZE);
}
hipError_t InitWin(int** dev_win, size_t POP_SIZE) {
	return hipMalloc(dev_win, sizeof(int) * POP_SIZE );
}

int cantInvalidosHost(float* fitness, size_t SIZE) {
	int total = 0;
	for (int i = 0; i < SIZE; i++) {
		total += fitness[i] < 0 ? 1 : 0;
	}
	return total;
}

void printInfo(int gen, const EvalInfo& eval) {
	printf("gen %d: Inval: %d, AvgP: %.2f, MinV: %.0f, AvgV: %.1f, Max: %.0f\n", gen, eval.invalidos, eval.avgPenal, eval.minValido, eval.avgValido, eval.max);
}

float* mem1;
void initMemory() {
	hipHostMalloc(&mem1, sizeof(float));
}

// evaluate comun

ErrorInfo evaluate_(size_t POP_SIZE, float* dev_fit, EvalInfo& eval,int gen) {

	bool mostrar = (SALIDA && (gen % SALIDA_STEP) == 0);
	ErrorInfo status;
	float avgFit, avgFitVal, minFitVal;
	T_FIT minFit, maxFit;
	float cantInv;
	float host_stats[3];
	size_t STAT_SIZE = sizeof(float) * 3;
	//hipHostMalloc(&host_stats, STAT_SIZE);

	status.cuda = hipGetLastError();
	if (status.failed()) return status;




	float* out1;
	float* out3;

	int nroBlocks = (POP_SIZE + MAX_THREADS_PER_BLOCK - 1) / (MAX_THREADS_PER_BLOCK);
	hipMalloc(&out1, nroBlocks * sizeof(float));
	hipMalloc(&out3, STAT_SIZE);

	

	// hallar maximo (se calcula siempre para conocer mejor iteracion)
	maximo << <nroBlocks, MAX_THREADS_PER_BLOCK >> >(dev_fit, out1, POP_SIZE);
	maximo << <1, MAX_THREADS_PER_BLOCK >> >(out1, out1, nroBlocks);
	// para copias suficientemente peque�as no se observo mejor performance en usar memoria pinned
	hipMemcpy(&maxFit, out1, sizeof(T_FIT), hipMemcpyDeviceToHost); //hipMemcpy(mem1, out1, sizeof(T_FIT), hipMemcpyDeviceToHost);
	
	eval.max = maxFit; 
	

	// hallar minimo
	/*minimo << <nroBlocks, MAX_THREADS_PER_BLOCK >> >(dev_fit, out1, POP_SIZE);
	minimo << <1, MAX_THREADS_PER_BLOCK >> >(out1, out1, nroBlocks);
	hipMemcpy(&minFit, out1, sizeof(T_FIT), hipMemcpyDeviceToHost);*/

	// promedio
	/*sumar << <nroBlocks, MAX_THREADS_PER_BLOCK >> >(dev_fit, out1, POP_SIZE);
	sumar << <1, MAX_THREADS_PER_BLOCK >> >(out1, out1, nroBlocks);
	hipMemcpy(&avgFit, out2, sizeof(float), hipMemcpyDeviceToHost);*/

	// calcular otras estadisticas solo si se muestran
	if (mostrar) {
		// cant invalidos, promedio validos e invalidos
		sumarValidez << <1, MAX_THREADS_PER_BLOCK >> >(dev_fit, out3, POP_SIZE);
		hipMemcpy(&host_stats, out3, STAT_SIZE, hipMemcpyDeviceToHost);
		eval.invalidos = host_stats[0];
		eval.avgPenal = eval.invalidos > 0 ? host_stats[1] / eval.invalidos : 0;
		eval.avgValido = eval.invalidos < POP_SIZE ? host_stats[2] / (POP_SIZE - eval.invalidos) : 0;

		// minimo validos
		minimoValidos << <1, MAX_THREADS_PER_BLOCK >> >(dev_fit, out1, POP_SIZE);
		hipMemcpy(&minFitVal, out1, sizeof(float), hipMemcpyDeviceToHost);
		eval.minValido = minFitVal;
	}

	status.cuda = hipGetLastError();
	if (status.failed()) return status;

	status.cuda = hipDeviceSynchronize();
	if (status.failed()) return status;


	hipFree(out1);
	hipFree(out3);


	if (mostrar) printInfo(gen, eval);

	return status;

}


ErrorInfo evaluate(bool* pop, size_t POP_SIZE, int length, float* dev_fit, EvalInfo& eval, float* W, float* G, int gen, float MAX_WEIGHT) {

	hipEvent_t startFitness, stopFitness;
	if (KERNEL_TIMING) {
		hipEventCreate(&startFitness);
		hipEventCreate(&stopFitness);

		hipEventRecord(startFitness);
	} // KERNEL_TIMING

	fitness_knapsack << < POP_SIZE, MAX_THREADS_PER_BLOCK >> > (pop, dev_fit, length, W, G, MAX_WEIGHT, PENAL);
	hipDeviceSynchronize();
	if (KERNEL_TIMING) {
		hipEventRecord(stopFitness);
		hipEventSynchronize(stopFitness);
		float milisecsFitness = 0;
		hipEventElapsedTime(&milisecsFitness, startFitness, stopFitness);
		timeFitness += milisecsFitness;
	}  // KERNEL_TIMING

	return evaluate_(POP_SIZE, dev_fit, eval,gen);
}


ErrorInfo evaluate_bitwise(Data* pop, size_t POP_SIZE, int realLength,int length, float* dev_fit, EvalInfo& eval,float* W, float* G,int gen,float MAX_WEIGHT) {
	hipEvent_t startFitness, stopFitness;
	if (KERNEL_TIMING) {
		hipEventCreate(&startFitness);
		hipEventCreate(&stopFitness);

		hipEventRecord(startFitness);
	}  // KERNEL_TIMING
	fitness_knapsack_b << < POP_SIZE, MAX_THREADS_PER_BLOCK >> > (pop, dev_fit, length,realLength, FirstBitMask, W, G, MAX_WEIGHT, PENAL);
	hipDeviceSynchronize();
	if (KERNEL_TIMING) {
		hipEventRecord(stopFitness);
		hipEventSynchronize(stopFitness);

		float milisecsFitness = 0;
		hipEventElapsedTime(&milisecsFitness, startFitness, stopFitness);
		timeFitness += milisecsFitness;
	}
	return evaluate_(POP_SIZE, dev_fit, eval,gen);

}


// Thamas Wang
// http://www.burtleburtle.net/bob/hash/integer.html
uint64_t hash64shift(uint64_t key)
{
	key = (~key) + (key << 21); // key = (key << 21) - key - 1;
	key = key ^ (key >>  24);
	key = (key + (key << 3)) + (key << 8); // key * 265
	key = key ^ (key >>  14);
	key = (key + (key << 2)) + (key << 4); // key * 21
	key = key ^ (key >>  28);
	key = key + (key << 31);
	return key;
}

/*__global__ void initPop_device(bool *pop, unsigned int length,unsigned long long seed) {
	unsigned int thIdx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandStatePhilox4_32_10_t rndState;
	hiprand_init(seed + thIdx, 0ull, 0ull, &rndState);
	for (unsigned int i = threadIdx.x; i < length; i = i + INIT_THREADS) {
		unsigned int pos = blockIdx.x * length + i;
		pop[pos] = (hiprand_uniform(&rndState) <= 0.5);
	}
}*/



/*__global__ void initPop_device32(bool *pop, unsigned int length, unsigned long long seed) {
	unsigned int thIdx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandStatePhilox4_32_10_t rndState;
	hiprand_init(seed + thIdx, 0ull, 0ull, &rndState);
	for (unsigned int i = threadIdx.x; i < length; ) {
		uint32_t rnd = hiprand(&rndState);
		for (uint32_t j = 0; j < 32 & i < length; j++, i = i + INIT_THREADS) {
			unsigned int pos = blockIdx.x * length + i;
			pop[pos] = (rnd & (1 << j)) != 0;
		}

	}
}*/







__global__ void WG_Fijos(float* W, float* G, int len) {
	unsigned int thIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thIdx < len) {
		W[thIdx] = thIdx % 2 + 1;
		G[thIdx] = thIdx % 10;
	}

}


void inicializarWG(float** W, float** G, int len) {
	hipMalloc(W, len * sizeof(float));
	hipMalloc(G, len * sizeof(float));
	int blocks = ceil(len / MAX_THREADS_PER_BLOCK);
	WG_Fijos << <blocks, MAX_THREADS_PER_BLOCK >> > (*W, *G, len);


}

void generarAleatorioPacket(hiprandGenerator_t& generator, size_t bytes, void* buffer) {
	size_t N = (3 + bytes) / 4;
	unsigned int *ptr = reinterpret_cast<unsigned int*>(buffer);
	hiprandGenerate(generator, ptr, N);
}



ErrorInfo GA(size_t POP_SIZE,int len,int iters,bool dpx_cross,float crossProb,float mutProb,	unsigned long long seed, float MAX_WEIGHT) {
	/*hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);*/

	int NRO_BLOCKS = (POP_SIZE + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
	double max_fitness;
	int gen_max_fitness = 0;
	if (dpx_cross) printf("DPX ");
	else printf("SPX ");

	printf("bool POP_SIZE=%u length=%d seed=%u pCross=%.2f pMut=%.2f\n",  POP_SIZE, len, seed, crossProb, mutProb);
	ErrorInfo status;
	bool *pop, *npop;
	float* fit;
	int* win; // indices de individuos ganadores en el tournment
	int* tourn;
	float  *probs;
	int *points;


	float *W;
	float *G;
	inicializarWG(&W, &G, len);

	EvalInfo eval;
	status.cuda = InitFit(&fit, POP_SIZE);
	status.cuda = InitWin(&win, POP_SIZE);
	status.cuda = InitTournRandom(&tourn, POP_SIZE);
	status = initProbs(&probs, &points, POP_SIZE);

	hiprandGenerator_t generator;
	status.hiprand = initGenerator(generator, seed);
	if (status.failed()) return status;


	//status = generatePOP(generator, POP_SIZE, len, &pop,&npop);
	// usa la hiprand device API para generar la poblacion sin prealocar numeros aleatorios para eso
	status = generatePOP_device(seed, POP_SIZE, len, &pop, &npop);
	//status = generatePOP_device(hash64shift(seed), POP_SIZE, len, &pop, &npop);

	// cambia el offset del generador para que no se sobreponga con el usado para la generacion de la poblacion
	hiprandSetGeneratorOffset(generator, POP_SIZE * len);

	if (status.failed()) {
		fprintf(stderr, "generatePOP failed!");
		return status;
	}

	int gen = 0;
	status = evaluate(pop, POP_SIZE, len,fit,eval,W,G,gen,MAX_WEIGHT);
	max_fitness = eval.max;


	for ( gen = 1; gen <= iters; gen++) { // while not optimalSolutionFound
		// elegir POP_SIZE parejas para el torneo
		status = makeRandomNumbersTournement(generator, POP_SIZE, tourn);
		if (status.failed()) return status;

		// elegir POP_SIZE ganadores
		tournament<<< NRO_BLOCKS , MAX_THREADS_PER_BLOCK>>> (fit, tourn, win,POP_SIZE);
		status.cuda = hipGetLastError();
		if (status.failed()) return status;

		// seleccion
		if (dpx_cross) {
			makeRandomNumbersDpx(generator, POP_SIZE, len, probs, points);
		}
		else {
			makeRandomNumbersSpx(generator, POP_SIZE, len, probs, points);
		}

		hipEvent_t startCross, stopCross;
		if (KERNEL_TIMING) {
			hipEventCreate(&startCross);
			hipEventCreate(&stopCross);

			hipEventRecord(startCross);
		} 	 // KERNEL_TIMING
		
		if (dpx_cross){
			dpx << < POP_SIZE / 2, MAX_THREADS_PER_BLOCK >> >(pop, npop, win, probs, points, len, crossProb);
		}
		else {
			spx << < POP_SIZE / 2, MAX_THREADS_PER_BLOCK >> >(pop, npop, win, probs, points, len, crossProb);
		}
		status.cuda = hipGetLastError();
		if (status.failed()) return status;

		status.cuda = hipDeviceSynchronize();
		if (status.failed()) return status;
		if (KERNEL_TIMING) {
			hipEventRecord(stopCross);
			hipEventSynchronize(stopCross);
			float milisecs = 0;
			hipEventElapsedTime(&milisecs, startCross, stopCross);
			timeCross += milisecs;
		} // KERNEL_TIMING


		// elegir numeros aleatorios para mutacion 
		// se reusa la memoria que se uso para los numeros aleatorios de la seleccion
		status = makeRandomNumbersMutation(generator, POP_SIZE, len, probs, points);

		// mutacion
		mutation << < NRO_BLOCKS, MAX_THREADS_PER_BLOCK >> >(npop, probs, points, len, mutProb,POP_SIZE);
		status.cuda = hipGetLastError();
		if (status.failed()) return status;
		status.cuda = hipDeviceSynchronize();

		bool* tmp;
		tmp = pop;
		pop = npop;
		npop = tmp;
		status = evaluate(pop, POP_SIZE, len, fit, eval, W, G,gen, MAX_WEIGHT);
		
		if (eval.max > max_fitness) {
			gen_max_fitness = gen;
			max_fitness = eval.max;
		}
	}
	printf("Gen. max fitness: %d (%f)\n", gen_max_fitness, max_fitness);
	return status;
}

ErrorInfo GA_bitwise(size_t POP_SIZE, int len, int iters, bool dpx_cross, float crossProb, float mutProb,unsigned long long seed, float MAX_WEIGHT) {

	int NRO_BLOCKS = (POP_SIZE + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
	if (dpx_cross) printf("DPX "); 
	else printf("SPX ");
	printf("bitwise(%u) POP_SIZE=%u length=%d seed=%u pCross=%.2f pMut=%.2f\n",sizeof(Data) * 8, POP_SIZE, len, seed,crossProb,mutProb);
	ErrorInfo status;
	Data *pop, *npop;
	float* fit;
	int* win;  // indices de individuos ganadores en el tournment
	int* tourn;
	float  *probs;
	int *points;

	double max_fitness;
	int gen_max_fitness = 0;

	float *W;
	float *G;
	inicializarWG(&W, &G, len);


	EvalInfo eval;
	int realLength = (len + DataSize - 1) / DataSize;
	status.cuda = InitFit(&fit, POP_SIZE);
	status.cuda = InitWin(&win, POP_SIZE);
	status.cuda = InitTournRandom(&tourn, POP_SIZE);
	status = initProbs(&probs, &points, POP_SIZE);


	hiprandGenerator_t generator;
	status.hiprand = initGenerator(generator, seed);
	if (status.failed()) return status;


	//status = generatePOP(generator, POP_SIZE, len, &pop,&npop);
	// usa la hiprand device API para generar la poblacion sin prealocar numeros aleatorios para eso
	status = generatePOP_device_bitwise(seed, POP_SIZE, len, &pop, &npop);

	//generarAleatorioPacket(generator, realLength * DataSize / 8, (void*)pop);
	



	// cambia el offset del generador para que no se sobreponga con el usado para la generacion de la poblacion
	hiprandSetGeneratorOffset(generator, POP_SIZE * len);

	if (status.failed()) {
		fprintf(stderr, "generatePOP failed!");
		return status;
	}

	int gen = 0;
	status = evaluate_bitwise(pop, POP_SIZE, realLength,len, fit, eval,W,G,gen, MAX_WEIGHT);
	max_fitness = eval.max;

	for ( gen = 1; gen <= iters; gen++) { // while not optimalSolutionFound
											 // elegir POP_SIZE parejas para el torneo
		status = makeRandomNumbersTournement(generator, POP_SIZE, tourn);
		if (status.failed()) return status;

		// elegir POP_SIZE ganadores
		tournament << < NRO_BLOCKS, MAX_THREADS_PER_BLOCK >> > (fit, tourn, win,POP_SIZE);
		status.cuda = hipGetLastError();
		if (status.failed()) return status;

		// seleccion
		if (dpx_cross) {
			makeRandomNumbersDpx(generator, POP_SIZE, len, probs, points);
		}
		else {
			makeRandomNumbersSpx(generator, POP_SIZE, len, probs, points);
		}
		hipEvent_t startCross, stopCross;
		if (KERNEL_TIMING) {
			
			hipEventCreate(&startCross);
			hipEventCreate(&stopCross);

			hipEventRecord(startCross);
		} //KERNEL_TIMING
		if (dpx_cross) {
			dpx_b << < POP_SIZE / 2, MAX_THREADS_PER_BLOCK >> >(pop, npop, win, probs, points, realLength, crossProb);
		}
		else {
			spx_b << < POP_SIZE / 2, MAX_THREADS_PER_BLOCK >> >(pop, npop, win, probs, points, realLength, crossProb);
		}
		status.cuda = hipGetLastError();
		if (status.failed()) return status;

		status.cuda = hipDeviceSynchronize();
		if (status.failed()) return status;
		if (KERNEL_TIMING) {
			hipEventRecord(stopCross);
			hipEventSynchronize(stopCross);
			float milisecs = 0;
			hipEventElapsedTime(&milisecs, startCross, stopCross);
			timeCross += milisecs;
		} //KERNEL_TIMING

		// elegir numeros aleatorios para mutacion 
		// se reusa la memoria que se uso para los numeros aleatorios de la seleccion
		status = makeRandomNumbersMutation(generator, POP_SIZE, len, probs, points);

		// mutacion
		mutation_b << < NRO_BLOCKS, MAX_THREADS_PER_BLOCK >> >(npop, probs, points, realLength, FirstBitMask, mutProb,POP_SIZE);
		status.cuda = hipGetLastError();
		if (status.failed()) return status;
		status.cuda = hipDeviceSynchronize();

		Data* tmp;
		tmp = pop;
		pop = npop;
		npop = tmp;
		status  = evaluate_bitwise(pop, POP_SIZE, realLength, len, fit, eval, W, G,gen, MAX_WEIGHT);

		if (eval.max > max_fitness) {
			gen_max_fitness = gen;
			max_fitness = eval.max;
		}
	}
	printf("Gen. max fitness: %d (%f)\n", gen_max_fitness, max_fitness);
	return status;
}





void setArgumentsFromCmd(int argc, char** argv,float& pMutacion, float& pCruce, float& MAX_WEIGHT, unsigned int& POP_SIZE,int& length, int& iters, unsigned long long& seed,bool& use_dpx,bool& bitwise) {
	// valores por defecto
	POP_SIZE = 2048;
	length = 10000;
	iters = 10000;
	pMutacion = 0.4;
	pCruce = 1;
	seed = 2825521;
	use_dpx = false;
	bitwise = true;

	bool relativeW = true;
	float w = 0.1;
	for (int i = 1; i < argc; i++) {
		if (strcmp("-k", argv[i]) == 0) {
			KERNEL_TIMING = true; // -k = Activar KERNEL_TIMING 
		}
		if (strcmp("-dpx", argv[i]) == 0) {
			use_dpx = true;
		}
		if (strcmp("-u", argv[i]) == 0) {
			UNIT_TEST = true;
		}
		if (strcmp("-bool", argv[i]) == 0) {
			bitwise = false;
		}
		if (strcmp("-o", argv[i]) == 0) {
			if (i + 1 < argc) SALIDA_STEP = atoi(argv[i + 1]);
		}

		if (strcmp("-m", argv[i]) == 0) {
			if (i + 1 < argc) pMutacion = (float)(atof(argv[i + 1]) / 100.0); // porcentaje
		}
		if (strcmp("-x", argv[i]) == 0) {
			if (i + 1 < argc) pCruce = (float)(atof(argv[i + 1]) / 100.0);  // porcentaje
		}
		if (strcmp("-w", argv[i]) == 0) {
			if (i + 1 < argc) w = (float)(atof(argv[i + 1]) ); // maximo peso = proporcion de longitud
		}
		if (strcmp("-W", argv[i]) == 0) {
			if (i + 1 < argc) {
				w = (float)(atof(argv[i + 1])); // maximo peso absoluto (ignora -w)
				relativeW = false;
			}
		}
		if (strcmp("-len", argv[i]) == 0) {
			if (i + 1 < argc) length = (atoi(argv[i + 1])); // longitud del individuo
		}
		if (strcmp("-p", argv[i]) == 0) {
			if (i + 1 < argc) POP_SIZE = (unsigned int)(atoi(argv[i + 1])); // longitud del individuo
		}
		if (strcmp("-i", argv[i]) == 0) {
			if (i + 1 < argc) iters = (atoi(argv[i + 1])); // iteraciones
		}
		if (strcmp("-s", argv[i]) == 0) {
			if (i + 1 < argc) seed = (unsigned long long)atoll(argv[i + 1]);
		}

	}
	if (relativeW) {
		MAX_WEIGHT = w * length;
	}
}


// A utility function that returns maximum of two integers
int max_(int a, int b) { return (a > b) ? a : b; }

// http://www.geeksforgeeks.org/knapsack-problem/
// Returns the maximum value that can be put in a knapsack of capacity W
int knapSack(int W, int wt[], int val[], int n)
{
	int i, w;
	int** K = new int*[n + 1];
	for (int i = 0; i <= n; i++) {
		K[i] = new int[W + 1];
	}

	// Build table K[][] in bottom up manner
	for (i = 0; i <= n; i++)
	{
		for (w = 0; w <= W; w++)
		{
			if (i == 0 || w == 0)
				K[i][w] = 0;
			else if (wt[i - 1] <= w)
				K[i][w] = max_(val[i - 1] + K[i - 1][w - wt[i - 1]], K[i - 1][w]);
			else
				K[i][w] = K[i - 1][w];
		}
	}

	int res =  K[n][W];
	for (int i = 0; i <= n; i++) {
		delete[] K[i];
	}
	delete[] K;
	return res;
}

__global__ void initArrays(int combinations,int len, int realLength, bool* pop_bool, Data* pop_bw,int* pos, float* randomPC,int* randomPoint) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < len) {
		pop_bool[idx] = false;
		pop_bool[idx + len] = true;
	}
	if (idx < realLength) {
		pop_bw[idx] = 0;
		pop_bw[idx + realLength] = ~((Data)0);
	}
	if (idx < combinations) {
		randomPC[ idx] = 0;
		randomPoint[2 * idx] = idx / len;
		randomPoint[2 * idx + 1] = idx % len;
		pos[2 * idx] = 0;
		pos[2 * idx + 1] = 1;
	}
}

__global__ void comparar(Data* npop, bool* npop_bool, int* output, int length,int realLength) {
	Data aux;
	const int tid = threadIdx.x;
	__shared__ int intermedio[MAX_THREADS_PER_BLOCK];
	intermedio[tid] = 0;
	for (int i = tid;i < length;i = i + MAX_THREADS_PER_BLOCK) {
		int k = i / DataSize;
		int j = (DataSize - 1) - i % DataSize;

		aux = npop[blockIdx.x*realLength + k];
		bool value_bit = ((aux >> j) & 1) != 0;
		bool value_bool = npop_bool[blockIdx.x * length + i];
		if (value_bit != value_bool) {
			intermedio[tid] += 1;
		}
	}
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s != 0; s >>= 1) {
		if (tid < s) {
			intermedio[tid] =intermedio[tid] + intermedio[tid + s];
		}
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		output[blockIdx.x] = intermedio[0];
	}

	

}
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	return;
}

inline void __cudaCheckError(const char *file, const int line)
{

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	// More careful checking. However, this will affect performance.
	// Comment away if needed.
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	return;
}

void PU_DPX() {
	Data *pop, *npop;
	bool *pop_bool, *npop_bool;

	Data *host_npop_bw;
	bool *host_npop_bool;
	
	float* randomPC;
	int *randomPoint, *pos, *output, *host_output;

	int largo = 64;
	int realLength = ((largo + DataSize - 1) / DataSize);
	int D = realLength * sizeof(Data);
	int output_count = 2 * largo * largo;
	hipMalloc(&pop, 2 * D );
	CudaCheckError();
	hipMalloc(&npop, output_count * D);
	CudaCheckError();
	hipMalloc(&pop_bool, 2 * largo * sizeof(bool));
	hipMalloc(&npop_bool, output_count * largo *  sizeof(bool));
	CudaCheckError();

	hipMalloc(&pos, output_count * sizeof(int));
	CudaCheckError();
	hipMalloc(&randomPoint, output_count * sizeof(int));
	CudaCheckError();
	hipMalloc(&randomPC, output_count * sizeof(float));
	CudaCheckError();
	hipMemset(randomPC, 0, output_count * sizeof(float));
	CudaCheckError();

	hipMalloc(&output, output_count * sizeof(int));
	CudaCheckError();
	hipHostMalloc(&host_output, output_count * sizeof(int));
	CudaCheckError();

	initArrays<<<largo, largo >>>(largo * largo, largo, realLength, pop_bool,pop, pos,randomPC, randomPoint);
	CudaCheckError();

	dpx << <output_count /2, MAX_THREADS_PER_BLOCK >> >(pop_bool, npop_bool, pos,randomPC, randomPoint, largo, 1);
	CudaCheckError();
	dpx_b << <output_count /2, MAX_THREADS_PER_BLOCK >> >(pop, npop,pos, randomPC, randomPoint, realLength, 1);
	CudaCheckError();
	comparar<<<output_count, MAX_THREADS_PER_BLOCK>>>(npop, npop_bool, output, largo, realLength);
	CudaCheckError();
	hipMemcpy(host_output, output, output_count, hipMemcpyDeviceToHost);
	CudaCheckError();

	hipHostMalloc(&host_npop_bool, output_count * largo *  sizeof(bool));
	hipHostMalloc(&host_npop_bw,  output_count * D);
	hipMemcpy(host_npop_bool, npop_bool, output_count * largo *  sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(host_npop_bw, npop, output_count * D, hipMemcpyDeviceToHost);

	for (int i = 0; i < output_count; i++) {
		if (host_output[i] > 0 | true ) {
			int pnt1 = (i / 2) / largo;
			int pnt2 = (i / 2) % largo;
			printf("%d: %d,%d[%d] -> err=%d\n", i, pnt1, pnt2, i % 2, host_output[i]);
			printf("bool: ");
			for (int j = 0; j < largo; j++) {
				printf("%d", host_npop_bool[i* largo + j] ? 1 : 0);
			}
			printf("\n");
			printf("bw  : ");
			for (int j = 0; j < largo; j++) {
				
				int k1 = j / DataSize;
				int k2  = (DataSize - 1) - j % DataSize;
				Data aux  = host_npop_bw[i*realLength + k1];
				int value = (aux >> k2) & 1;
				printf("%d", value);
			}
			printf("\n");
		}
	}

}


int main(int argc, char** argv)
{


	hipError_t cudaStatus;
	ErrorInfo status;



	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	
	std::clock_t c_start = std::clock();

	timeFitness = timeCross = 0;

	// TODO: arreglar para POP_SIZE no multiplo de MAX_THREADS
	unsigned int POP_SIZE = 2048;
	 // peso maximo de la mochila
	int len,iters ;
	float MAX_WEIGHT, pMutacion, pCruce;
	unsigned long long seed ;
	bool use_dpx,bitwise ;

	setArgumentsFromCmd(argc, argv, pMutacion, pCruce, MAX_WEIGHT, POP_SIZE, len, iters, seed, use_dpx,bitwise);

	if (UNIT_TEST) {
		PU_DPX();
		exit(0);
	}

	if (bitwise) {
		GA_bitwise(POP_SIZE, len, iters, use_dpx, pCruce, pMutacion, seed, MAX_WEIGHT);
	}
	else {
		GA(POP_SIZE, len, iters, use_dpx, pCruce, pMutacion, seed, MAX_WEIGHT);
	}
	
	std::clock_t c_end = std::clock();
	double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	printf("Tiempo total: %.3fs\n", time_elapsed_ms / 1000.0);
	if (KERNEL_TIMING) {
		printf("Tiempo fitness: %.3fs, Tiempo cross: %.3fs\n", timeFitness / 1000, timeCross / 1000);
	}  //KERNEL_TIMING 



	//std::cout << "Press any key to exit . . .";
	//std::cin.get();
	return 0;
}


